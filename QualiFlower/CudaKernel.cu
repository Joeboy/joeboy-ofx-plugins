
#include <hip/hip_runtime.h>
__device__ void rgb2hslcuda(double r, double g, double b, double *h, double *s, double *l)
{
  // Copied from the CPU version, dunno if it's optimal for GPU
  // rgb are 0->1, return hsl as 0->100
  double min, max, delta;

  min = r < g ? r : g;
  min = min  < b ? min : b;

  max = r > g ? r : g;
  max = max  > b ? max  : b;

  // I seems these rgb values can be > 1. I'm going to just clamp them,
  // but maybe the right thing would be to assume a range of 0->highest_value_in_image,
  // rather than 1->1? Not sure.
  *l = max >= 1.0 ? 100.0 : 100.0 * max;

  delta = max - min;
  if (delta < 0.00001)
  {
      *s = 0;
      *h = 0;
      return;
  }
  if( max > 0.0 ) { // NOTE: if Max is == 0, this divide would cause a crash
      *s = 100 * (delta / max);
  } else {
      *s = 0.0;
      *h = NAN;
      return;
  }
  if( r >= max ) {                 // > is bogus, just keeps compiler happy
      *h = (g - b) / delta;        // between yellow & magenta
  } else if(g >= max) {
      *h = 2.0 + (b - r) / delta;  // between cyan & yellow
  } else {
      *h = 4.0 + (r - g) / delta;  // between magenta & cyan
  }

//    *h *= 60.0;                              // degrees
//    if(*h < 0.0 ) *h += 360.0;
  *h *= 100.0 / 6.0;
  if (*h < 0.0) *h += 100.0;

  return;
}


__global__ void HSLSelectKernel(
    int p_Width, int p_Height,
    bool _hueEnabled, float _hue, float _hueWidth, float _hueSoftness,
    bool _saturationEnabled, float _saturationLow, float _saturationHigh, float _saturationLowSoftness, float _saturationHighSoftness,
    bool _luminanceEnabled, float _luminanceLow, float _luminanceHigh, float _luminanceLowSoftness, float _luminanceHighSoftness,
    const float* p_Input, float* p_Output)
{
    // rgb are 0->1, return hsl as 0->100
    // Copied from the CPU version, dunno if it's optimal for GPU
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    float r, g, b;
    double h, s, l;
    double minHue, maxHue, overflowed_h, underflowed_h;
    double hue_multiplier, sat_multiplier, lum_multiplier;
    double hue_lower_softness_threshold, hue_upper_softness_threshold;
    minHue = _hue - .5 * _hueWidth;
    maxHue = _hue + .5 * _hueWidth;
    hue_lower_softness_threshold = minHue - _hueSoftness;
    hue_upper_softness_threshold = maxHue + _hueSoftness;

   if ((x < p_Width) && (y < p_Height))
   {
        const int index = ((y * p_Width) + x) * 4;
        r = p_Input[index + 0];
        g = p_Input[index + 1];
        b = p_Input[index + 2];
        rgb2hslcuda(
            r, g, b,
            &h, &s, &l
        );


        if (_hueEnabled) {
            overflowed_h = h - 100.0;  // "wrapped around" hue, for testing against negative softness window
            underflowed_h = h + 100.0; // "wrapped around" hue, for testing against overflowed softness window
            if (h >= minHue && h <= maxHue) {
                hue_multiplier = 1.0;
            } else if (overflowed_h >= minHue && overflowed_h <= maxHue) {
                hue_multiplier = 1.0;
            } else if (underflowed_h >= minHue && underflowed_h <= maxHue) {
                hue_multiplier = 1.0;
            } else if (h > hue_lower_softness_threshold && h < minHue) {
                hue_multiplier = (h - hue_lower_softness_threshold) / _hueSoftness;
            } else if (overflowed_h > hue_lower_softness_threshold && overflowed_h < minHue) {
                hue_multiplier = (overflowed_h - hue_lower_softness_threshold) / _hueSoftness;
            } else if (h > maxHue && h <= hue_upper_softness_threshold) {
                hue_multiplier = (hue_upper_softness_threshold - h) / _hueSoftness;
            } else if (underflowed_h > maxHue && underflowed_h <= hue_upper_softness_threshold) {
                hue_multiplier = (hue_upper_softness_threshold - underflowed_h) / _hueSoftness;
            } else {
                hue_multiplier = 0.0;
            }
        } else hue_multiplier = 1.0;
        //if (cc<1) printf("%f %f %f min=%f max=%f a=%f\n", h, s, v, minHue, maxHue, a);
        //if (cc<1) printf("%f %f %f min=%f max=%f\n", h, s, l, luminance_low, luminance_high);

        if (_saturationEnabled) {
            if (s >= _saturationLow && s <= _saturationHigh) {
                sat_multiplier = 1.0;
            } else if (s < _saturationLow && s > _saturationLow - _saturationLowSoftness) {
                sat_multiplier = (s - (_saturationLow - _saturationLowSoftness)) / _saturationLowSoftness;
            } else if (s > _saturationHigh && s < _saturationHigh + _saturationHighSoftness){
                sat_multiplier = 1.0 - (s - _saturationHigh) / _saturationHighSoftness;
            } else {
                sat_multiplier = 0.0;
            }
        } else sat_multiplier = 1.0;

        if (_luminanceEnabled) {
            if (l >= _luminanceLow && l <= _luminanceHigh) {
                lum_multiplier = 1.0;
            } else if (l < _luminanceLow && l > _luminanceLow - _luminanceLowSoftness) {
                lum_multiplier = (l - (_luminanceLow - _luminanceLowSoftness)) / _luminanceLowSoftness;
            } else if (s > _luminanceHigh && l < _luminanceHigh + _luminanceHighSoftness){
                lum_multiplier = 1.0 - (l - _luminanceHigh) / _luminanceHighSoftness;
            } else {
                lum_multiplier = 0.0;
            }
        } else lum_multiplier = 1.0;
        //if (cc<1) printf("lum multiplier=%f\n", lum_multiplier);


        p_Output[index + 0] = r;
        p_Output[index + 1] = g;
        p_Output[index + 2] = b;
        p_Output[index + 3] = hue_multiplier * sat_multiplier * lum_multiplier;
    }
}

void RunCudaKernel(
    void* p_Stream, int p_Width, int p_Height,
    bool hueEnabled, float hue, float hueWidth, float hueSoftness,
    bool saturationEnabled, float saturationLow, float saturationHigh, float saturationLowSoftness, float saturationHighSoftness,
    bool luminanceEnabled, float luminanceLow, float luminanceHigh, float luminanceLowSoftness, float luminanceHighSoftness,
    const float* p_Input, float* p_Output)
{
    dim3 threads(128, 1, 1);
    dim3 blocks(((p_Width + threads.x - 1) / threads.x), p_Height, 1);
    hipStream_t stream = static_cast<hipStream_t>(p_Stream);

    HSLSelectKernel<<<blocks, threads, 0, stream>>>(
        p_Width, p_Height,
        hueEnabled, hue, hueWidth, hueSoftness,
        saturationEnabled, saturationLow, saturationHigh, saturationLowSoftness, saturationHighSoftness,
        luminanceEnabled, luminanceLow, luminanceHigh, luminanceLowSoftness, luminanceHighSoftness,
        p_Input, p_Output
    );
}

